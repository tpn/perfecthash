#include "hip/hip_runtime.h"
/*++

Copyright (c) 2020 Trent Nelson <trent@trent.me>

Module Name:

    Graph.cu

Abstract:

    CUDA graph implementation.

--*/

#define PH_CU

#include <PerfectHash.h>

EXTERN_C_BEGIN
#include "../PerfectHash/CuDeviceAttributes.h"
#include "../PerfectHash/Graph.h"
//#include "../PerfectHash/Cu.h"

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
EXTERN_C_END

#include <hiprand/hiprand_kernel.h>

#include <thrust/device_ptr.h>
#include <thrust/sort.h>
#include <thrust/inner_product.h>
#include <thrust/unique.h>

#include "Graph.cuh"

//
// Shared memory.
//

extern SHARED ULONG SharedRaw[];

EXTERN_C
DEVICE
VOID
PerfectHashPrintCuErrorGraphThrust(
    PCSZ FunctionName,
    PCSZ FileName,
    ULONG LineNumber,
    ULONG Error
    )
{
    PCSZ ErrorName;
    PCSZ ErrorString;

    ErrorName = hipGetErrorName((CU_RESULT)Error);
    ErrorString = hipGetErrorString((CU_RESULT)Error);

    //
    // Error message format:
    //
    //      <FileName>:<LineNumber>: <Name> failed with error <Code>: \
    //          <ErrorName>: <ErrorString>.
    //

    printf("%s:%d %s failed with error 0x%x: %s: %s.\n",
           FileName,
           LineNumber,
           FunctionName,
           Error,
           ErrorName,
           ErrorString);
}

EXTERN_C
DEVICE
VOID
PerfectHashPrintErrorGraphThrust(
    PCSZ FunctionName,
    PCSZ FileName,
    ULONG LineNumber,
    ULONG Result
    )
{
    printf("%s:%d %s failed with error 0x%x.\n",
           FileName,
           LineNumber,
           FunctionName,
           Result);
}

#undef PH_ERROR
#define PH_ERROR(Name, Result)                      \
    PerfectHashPrintErrorGraphThrust(#Name,         \
                                     __FILE__,      \
                                     __LINE__,      \
                                     (ULONG)Result)


#undef CU_ERROR
#define CU_ERROR(Name, CuResult)                 \
    PerfectHashPrintCuErrorGraphThrust(#Name,    \
                                       __FILE__, \
                                       __LINE__, \
                                       CuResult)



EXTERN_C
GLOBAL
VOID
HashAllMultiplyShiftRKernel(
    _In_reads_(NumberOfKeys) PKEY Keys,
    _In_ ULONG NumberOfEdges,
    _In_ ULONG NumberOfKeys,
    _Out_writes_(NumberOfKeys) PVERTEX_PAIR VertexPairs,
    _Out_writes_(NumberOfKeys) PVERTEX_PAIR SortedVertexPairs,
    _Out_writes_(NumberOfKeys) PULONG VertexPairsIndex,
    _Out_ PVERTEX First,
    _Out_ PEDGE Next,
    _Out_ PEDGE Edges,
    _In_ ULONG Mask,
    _In_ PULONG Seeds,
    _Out_ PHRESULT GlobalResult
    )
{
    KEY Key;
    EDGE Edge1;
    EDGE Edge2;
    EDGE First1;
    EDGE First2;
    ULONG Vertex1;
    ULONG Vertex2;
    ULONG Index;
    ULONG Seed1;
    ULONG Seed2;
    ULONG_BYTES Seed3;
    //PHRESULT BlockResult;
    //PGRAPH_SHARED Shared = (PGRAPH_SHARED)SharedRaw;

    //
    // Initialize aliases.
    //

    Seed1 = Seeds[0];
    Seed2 = Seeds[1];
    Seed3.AsULong = Seeds[2];

#if 0

    //
    // If this is thread 0 in the block, initialize the shared memory and set
    // the global result to S_OK.
    //

    if (ThreadIndex.x == 0) {

        Shared->HashKeysBlockResults = (PHRESULT)(
            RtlOffsetToPointer(
                SharedRaw,
                sizeof(GRAPH_SHARED)
            )
        );


        *GlobalResult = S_OK;
    }

    __syncthreads();

    BlockResult = &Shared->HashKeysBlockResults[BlockIndex.x];

#endif

    FOR_EACH_1D(Index, NumberOfKeys) {

#if 0
        //
        // Block-level fast-path exit if we've already detected a vertex
        // collision.  I haven't profiled things to determine if it makes
        // sense to either do: a) this, or b) an additional global memory
        // read of `*GlobalResult` (currently not being done).
        //

        if (*BlockResult != S_OK) {
            goto End;
        }
#endif

        Key = Keys[Index];

        Vertex1 = (((Key * SEED1) >> SEED3_BYTE1) & Mask);
        Vertex2 = (((Key * SEED2) >> SEED3_BYTE2) & Mask);

        if (Vertex1 == Vertex2) {

            //
            // Set the block-level and global-level results to indicate
            // collision, then jump to the end.
            //

            //*BlockResult = PH_E_GRAPH_VERTEX_COLLISION_FAILURE;
            *GlobalResult = PH_E_GRAPH_VERTEX_COLLISION_FAILURE;
            goto End;
        }

        //
        // Store the vertex pairs.
        //

        VertexPairs[Index].Vertex1 = Vertex1;
        VertexPairs[Index].Vertex2 = Vertex2;
        SortedVertexPairs[Index].Vertex1 = Vertex1;
        SortedVertexPairs[Index].Vertex2 = Vertex2;
        VertexPairsIndex[Index] = Index;

        Edge1 = (EDGE)Index;
        Edge2 = Edge1 + NumberOfEdges;

        //
        // Insert the first edge.
        //

        First1 = First[Vertex1];
        Next[Edge1] = First1;
        First[Vertex1] = Edge1;
        Edges[Edge1] = Vertex2;

        //
        // Insert the second edge.
        //

        First2 = First[Vertex2];
        Next[Edge2] = First2;
        First[Vertex2] = Edge2;
        Edges[Edge2] = Vertex1;
    }

End:
    return;
}

EXTERN_C
GLOBAL
VOID
HashAllMultiplyShiftRKernel2(
    _In_reads_(NumberOfKeys) PKEY Keys,
    _In_ ULONG NumberOfKeys,
    _Out_writes_(NumberOfKeys) PVERTEX Vertices1,
    _Out_writes_(NumberOfKeys) PVERTEX Vertices2,
    _Out_writes_(NumberOfKeys) PULONG Vertices1Index,
    _In_ ULONG Mask,
    _In_ PULONG Seeds,
    _Out_ PHRESULT GlobalResult
    )
{
    KEY Key;
    ULONG Index;
    ULONG Seed1;
    ULONG Seed2;
    ULONG_BYTES Seed3;
    VERTEX Vertex1;
    VERTEX Vertex2;
    PHRESULT BlockResult;
    PGRAPH_SHARED Shared = (PGRAPH_SHARED)SharedRaw;

    //
    // Initialize aliases.
    //

    Seed1 = Seeds[0];
    Seed2 = Seeds[1];
    Seed3.AsULong = Seeds[2];

    //
    // If this is thread 0 in the block, initialize the shared memory and set
    // the global result to S_OK.
    //

    if (ThreadIndex.x == 0) {

        Shared->HashKeysBlockResults = (PHRESULT)(
            RtlOffsetToPointer(
                SharedRaw,
                sizeof(GRAPH_SHARED)
            )
        );


        *GlobalResult = S_OK;
    }

    __syncthreads();

    BlockResult = &Shared->HashKeysBlockResults[BlockIndex.x];

    FOR_EACH_1D(Index, NumberOfKeys) {

        //
        // Block-level fast-path exit if we've already detected a vertex
        // collision.  I haven't profiled things to determine if it makes
        // sense to either do: a) this, or b) an additional global memory
        // read of `*GlobalResult` (currently not being done).
        //

        if (*BlockResult != S_OK) {
            goto End;
        }

        Key = Keys[Index];

        Vertex1 = (((Key * SEED1) >> SEED3_BYTE1) & Mask);
        Vertex2 = (((Key * SEED2) >> SEED3_BYTE2) & Mask);

        if (Vertex1 == Vertex2) {

            //
            // Set the block-level and global-level results to indicate
            // collision, then jump to the end.
            //

            *BlockResult = PH_E_GRAPH_VERTEX_COLLISION_FAILURE;
            *GlobalResult = PH_E_GRAPH_VERTEX_COLLISION_FAILURE;
            goto End;
        }

        //
        // Store the vertex pairs.
        //

        //Output[Index].x = Vertex1;
        //Output[Index].y = Vertex2;
        Vertices1[Index] = Vertex1;
        Vertices2[Index] = Vertex2;
        Vertices1Index[Index] = Index;

    }

End:
    return;
}

EXTERN_C
GLOBAL
VOID
HashAllMultiplyShiftRKernel3(
    _In_reads_(NumberOfKeys) PKEY Keys,
    _In_ ULONG NumberOfKeys,
    _Out_writes_(NumberOfKeys) PVERTEX Vertices1,
    _Out_writes_(NumberOfKeys) PVERTEX Vertices2,
    _Out_writes_(NumberOfKeys) PVERTEX_PAIR VertexPairs,
    _Out_writes_(NumberOfKeys) PULONG Vertices1Index,
    _Out_writes_(NumberOfKeys) PULONG Vertices2Index,
    _Out_writes_(NumberOfKeys) PULONG VertexPairsIndex,
    _In_ ULONG Mask,
    _In_ PULONG Seeds,
    _Out_ PHRESULT GlobalResult
    )
{
    KEY Key;
    ULONG Index;
    ULONG Seed1;
    ULONG Seed2;
    ULONG_BYTES Seed3;
    VERTEX Vertex1;
    VERTEX Vertex2;
    //PHRESULT BlockResult;
    //PGRAPH_SHARED Shared = (PGRAPH_SHARED)SharedRaw;
    //PINT2 Output = (PINT2)VertexPairs;

    //
    // Initialize aliases.
    //

    Seed1 = Seeds[0];
    Seed2 = Seeds[1];
    Seed3.AsULong = Seeds[2];

#if 0
    //
    // If this is thread 0 in the block, initialize the shared memory and set
    // the global result to S_OK.
    //

    if (ThreadIndex.x == 0) {

        Shared->HashKeysBlockResults = (PHRESULT)(
            RtlOffsetToPointer(
                SharedRaw,
                sizeof(GRAPH_SHARED)
            )
        );


        *GlobalResult = S_OK;
    }

    __syncthreads();

    BlockResult = &Shared->HashKeysBlockResults[BlockIndex.x];
#endif

    FOR_EACH_1D(Index, NumberOfKeys) {

        //
        // Block-level fast-path exit if we've already detected a vertex
        // collision.  I haven't profiled things to determine if it makes
        // sense to either do: a) this, or b) an additional global memory
        // read of `*GlobalResult` (currently not being done).
        //

#if 0
        if (*BlockResult != S_OK) {
            goto End;
        }
#endif

        Key = Keys[Index];

        Vertex1 = (((Key * SEED1) >> SEED3_BYTE1) & Mask);
        Vertex2 = (((Key * SEED2) >> SEED3_BYTE2) & Mask);

        if (Vertex1 == Vertex2) {

            //
            // Set the block-level and global-level results to indicate
            // collision, then jump to the end.
            //

            //*BlockResult = PH_E_GRAPH_VERTEX_COLLISION_FAILURE;
            *GlobalResult = PH_E_GRAPH_VERTEX_COLLISION_FAILURE;
            goto End;
        }

        //
        // Store the vertex pairs.
        //

        //Output[Index].x = Vertex1;
        //Output[Index].y = Vertex2;
        VertexPairs[Index].Vertex1 = Vertex1;
        VertexPairs[Index].Vertex2 = Vertex2;
        Vertices1[Index] = Vertex1;
        Vertices2[Index] = Vertex2;
        Vertices1Index[Index] = Index;
        Vertices2Index[Index] = Index;
        VertexPairsIndex[Index] = Index;

    }

End:
    return;
}

KERNEL
VOID
GraphCuSortVertices1Kernel(
    _In_ PGRAPH Graph
    )
{
    ULONG UniqueCount;
    thrust::device_ptr<ULONG> Vertices1(Graph->Vertices1Device);
    thrust::device_ptr<ULONG> Index(Graph->Vertices1IndexDevice);

#if 0
    thrust::stable_sort_by_key(thrust::device,
                               Vertices1,
                               Vertices1 + Graph->NumberOfKeys,
                               Index);
#endif

    thrust::stable_sort(
        thrust::device,
        Vertices1,
        Vertices1 + Graph->NumberOfKeys
    );

    UniqueCount = thrust::inner_product(
        thrust::device,
        Vertices1,
        Vertices1 + Graph->NumberOfKeys - 1,
        Vertices1 + 1,
        ULONG(1),
        thrust::plus<ULONG>(),
        thrust::not_equal_to<ULONG>()
    );

    printf("Vertices1:\n\tUniqueCount: %u\n\tKeys: %u\n\t"
           "Seed1: %u\n\tSeed2: %u\n\tSeed3: %u\n",
           UniqueCount,
           Graph->NumberOfKeys,
           Graph->Seed1,
           Graph->Seed2,
           Graph->Seed3);

}

KERNEL
VOID
GraphCuSortVertices2Kernel(
    _In_ PGRAPH Graph
    )
{
    thrust::device_ptr<ULONG> Vertices2(Graph->Vertices2Device);
    thrust::device_ptr<ULONG> Index(Graph->Vertices2IndexDevice);

    thrust::stable_sort_by_key(thrust::device,
                               Vertices2,
                               Vertices2 + Graph->NumberOfKeys,
                               Index);
}

DEVICE
bool
VertexPairLessThan(
    const VERTEX_PAIR Left,
    const VERTEX_PAIR Right
    )
{
    if (Left.Vertex1 < Right.Vertex1) {
        return true;
    } else if (Left.Vertex1 == Right.Vertex1) {
        return (Left.Vertex2 < Right.Vertex2);
    } else {
        return false;
    }
}

DEVICE
bool
VertexPairNotEqual(
    const VERTEX_PAIR Left,
    const VERTEX_PAIR Right
    )
{
    return (
        Left.Vertex1 != Right.Vertex1 &&
        Left.Vertex2 != Right.Vertex2
    );
    //return (Left.AsULongLong != Right.AsULongLong);
}

DEVICE
bool
VertexPairEqual(
    const VERTEX_PAIR Left,
    const VERTEX_PAIR Right
    )
{
    return (
        Left.Vertex1 == Right.Vertex1 &&
        Left.Vertex2 == Right.Vertex2
    );
    //return (Left.AsULongLong == Right.AsULongLong);
}


/*
KERNEL
VOID
GraphCuSortVertexPairsKernel(
    _In_ PGRAPH Graph
    )
{
    ULONG UniqueCount;
    thrust::device_ptr<VERTEX_PAIR> VertexPairs(Graph->VertexPairs);
    thrust::device_ptr<VERTEX_PAIR> VertexPairsEnd;
    thrust::device_ptr<VERTEX_PAIR> EndUnique;
    thrust::device_ptr<ULONG> Index(Graph->VertexPairsIndex);

    thrust::stable_sort(
        thrust::device,
        VertexPairs,
        VertexPairs + Graph->NumberOfKeys,
        VertexPairLessThan
    );

    UniqueCount = thrust::inner_product(
        thrust::device,
        VertexPairs,
        VertexPairs + Graph->NumberOfKeys - 1,
        VertexPairs + 1,
        ULONG(1),
        thrust::plus<ULONG>(),
        VertexPairNotEqual
    );

    printf("VertexPair NE:\n\tUniqueCount: %u\n\tKeys: %u\n\t"
           "Seed1: %u\n\tSeed2: %u\n\tSeed3: %u\n",
           UniqueCount,
           Graph->NumberOfKeys,
           Graph->Seed1,
           Graph->Seed2,
           Graph->Seed3);

    UniqueCount = thrust::inner_product(
        thrust::device,
        VertexPairs,
        VertexPairs + Graph->NumberOfKeys - 1,
        VertexPairs + 1,
        ULONG(1),
        thrust::plus<ULONG>(),
        VertexPairEqual
    );

    printf("VertexPair EQ:\n\tUniqueCount: %u\n\tKeys: %u\n\t"
           "Seed1: %u\n\tSeed2: %u\n\tSeed3: %u\n",
           UniqueCount,
           Graph->NumberOfKeys,
           Graph->Seed1,
           Graph->Seed2,
           Graph->Seed3);

    VertexPairsEnd = VertexPairs + Graph->NumberOfKeys;
    EndUnique = thrust::unique(
        thrust::device,
        VertexPairs,
        VertexPairsEnd,
        VertexPairEqual
    );

    if (VertexPairsEnd != EndUnique) {
        printf("Not unique!\n");
    } else {
        printf("All unique!\n");
    }

    printf("Start: 0x%p, End: 0x%p, EndUnique: 0x%p.\n",
           VertexPairs,
           VertexPairsEnd,
           EndUnique);

    printf("Num element: %u.\n", (VertexPairsEnd - VertexPairs));
    printf("Num unique: %u.\n", (EndUnique - VertexPairs));

}
*/

KERNEL
VOID
GraphCuSortVertexPairsKernel(
    _In_ ULONG NumberOfKeys,
    _Inout_ thrust::device_ptr<VERTEX_PAIR> VertexPairs,
    _Inout_ thrust::device_ptr<ULONG> VertexPairsIndex
    )
{
    thrust::device_ptr<VERTEX_PAIR> VertexPairsEnd;

    VertexPairsEnd = VertexPairs + NumberOfKeys;

    thrust::sort_by_key(
        thrust::device,
        VertexPairs,
        VertexPairsEnd,
        VertexPairsIndex,
        VertexPairLessThan
    );
}

KERNEL
VOID
GraphCuIsAcyclicKernel(
    _In_ ULONG NumberOfKeys,
    _Inout_ thrust::device_ptr<VERTEX_PAIR> VertexPairs,
    _Out_ PHRESULT Result
    )
{
    thrust::device_ptr<VERTEX_PAIR> VertexPairsEnd;
    thrust::device_ptr<VERTEX_PAIR> VertexPairsEndUnique;

    VertexPairsEnd = VertexPairs + NumberOfKeys;

    thrust::sort(
        thrust::device,
        VertexPairs,
        VertexPairsEnd,
        VertexPairLessThan
    );

    VertexPairsEndUnique = thrust::unique(
        thrust::device,
        VertexPairs,
        VertexPairsEnd,
        VertexPairEqual
    );

    if (VertexPairsEndUnique != VertexPairsEnd) {
        *Result = PH_E_GRAPH_CYCLIC_FAILURE;
    }
}

KERNEL
VOID
GraphCuAddEdgesKernel(
    _In_ ULONG NumberOfEdges,
    _In_ ULONG NumberOfKeys,
    _In_reads_(NumberOfKeys) PVERTEX_PAIR VertexPairs,
    _Out_ PEDGE Edges,
    _Out_ PEDGE Next,
    _Out_ PVERTEX First
    )
{
    EDGE Edge1;
    EDGE Edge2;
    EDGE First1;
    EDGE First2;
    ULONG Index;
    PINT2 Input = (PINT2)VertexPairs;
    VERTEX Vertex1;
    VERTEX Vertex2;

    FOR_EACH_1D(Index, NumberOfKeys) {
        Vertex1 = Input[Index].x;
        Vertex2 = Input[Index].y;

        Edge1 = (EDGE)Index;
        Edge2 = Edge1 + NumberOfEdges;

        //
        // Insert the first edge.
        //

        First1 = First[Vertex1];
        Next[Edge1] = First1;
        First[Vertex1] = Edge1;
        Edges[Edge1] = Vertex2;

        //
        // Insert the second edge.
        //

        First2 = First[Vertex2];
        Next[Edge2] = First2;
        First[Vertex2] = Edge2;
        Edges[Edge2] = Vertex1;
    }
}

EXTERN_C
DEVICE
BOOLEAN
GraphCuShouldWeContinueTryingToSolve(
    _In_ PGRAPH Graph,
    _Out_ PHRESULT Result
    )
{
    ULONG Target;
    ULONG Timeout;
    ULONG ClockRate;
    BOOLEAN CheckTimeout;
    ULONGLONG Delta;
    ULONGLONG ThisClock;
    PCU_DEVICE_ATTRIBUTES Attributes;

    if (Graph->Attempt++ == 0) {
        Graph->CuStartClock = clock64();
    } else {
        ThisClock = clock64();
        Delta = ThisClock - Graph->CuStartClock;
        Attributes = (PCU_DEVICE_ATTRIBUTES)Graph->CuDeviceAttributes;
        ClockRate = Attributes->ClockRate;

        Graph->CuEndClock = ThisClock;
        Graph->CuCycles = Delta;
        Graph->CuElapsedMilliseconds = Delta / ClockRate;

        CheckTimeout = (
            Attributes->KernelExecTimeout > 0 ||
            AlwaysRespectCuKernelRuntimeLimit(Graph)
        );

        if (CheckTimeout) {

            if (Attributes->KernelExecTimeout > 0) {

                //
                // There's a kernel timeout for this device.  Convert it to
                // milliseconds, then use whatever is the smaller value between
                // it and the user-specified kernel runtime limit.
                //
                // N.B. We subtract 10 milliseconds just so we're not *too*
                //      close to the Windows-enforced driver timeout limit.
                //

                Timeout = (Attributes->KernelExecTimeout * 1000) - 10;

                Target = min(Timeout,
                             Graph->CuKernelRuntimeTargetInMilliseconds);
            } else {

                Target = Graph->CuKernelRuntimeTargetInMilliseconds;
            }

            if (Graph->CuElapsedMilliseconds >= Target) {
                *Result = PH_S_CU_KERNEL_RUNTIME_TARGET_REACHED;
                return FALSE;
            }
        }
    }

    *Result = S_OK;

    return TRUE;
    //return (Graph->Attempt <= 5);
}

EXTERN_C
DEVICE
HRESULT
GraphCuApplySeedMasks(
    _In_ PGRAPH Graph
    )
{
    BYTE Index;
    BYTE NumberOfSeeds;
    LONG Mask;
    ULONG NewSeed;
    PULONG Seed;
    PULONG Seeds;
    const LONG *Masks;

    if (!HasSeedMasks(Graph)) {

        //
        // No seed masks are available for this hash routine.
        //

        return S_FALSE;
    }

    //
    // Validation complete.  Loop through the masks and apply those with a value
    // greater than zero to the seed at the corresponding offset.
    //

    Seeds = &Graph->FirstSeed;
    Masks = &Graph->SeedMasks.Mask1;

    NumberOfSeeds = (BYTE)Graph->NumberOfSeeds;

    for (Index = 0; Index < NumberOfSeeds; Index++) {

        Mask = *Masks++;

        if (Mask != -1 && Mask != 0) {

            //
            // Valid mask found, apply it to the seed data at this slot.
            //

            Seed = Seeds + Index;
            NewSeed = *Seed & Mask;
            *Seed = NewSeed;
        }
    }

    return S_OK;
}

EXTERN_C
DEVICE
HRESULT
GraphCuLoadNewSeeds(
    _In_ PGRAPH Graph
    )
/*++

Routine Description:

    Loads new seed data for a graph instance.  This is called prior to each
    solving attempt.

Arguments:

    Graph - Supplies a pointer to the graph instance for which the new seed
        data will be loaded.

Return Value:

    S_OK - Success.

--*/
{
    BYTE Index;
    PULONG Seed;
    BYTE NumberOfSeeds;
    HRESULT Result;
    PCU_KERNEL_CONTEXT Ctx;
    hiprandStatePhilox4_32_10_t *State;

    if (HasUserSeeds(Graph)) {

        //
        // The user has supplied seeds, so skip the hiprand() calls.
        //

        goto End;
    }

    Ctx = Graph->CuKernelContext;
    State = &Ctx->RngState.Philox4;

    Seed = &Graph->FirstSeed;
    NumberOfSeeds = (BYTE)Graph->NumberOfSeeds;

    for (Index = 0; Index < NumberOfSeeds; Index++) {
        *Seed++ = hiprand(State);
    }

End:

    Result = S_OK;

    if (HasSeedMasks(Graph)) {
        Result = GraphCuApplySeedMasks(Graph);
    }

    return Result;
}

EXTERN_C
DEVICE
HRESULT
GraphCuReset(
    _In_ PGRAPH Graph
    )
/*++

Routine Description:

    Resets the state of a graph instance after a solving attempt, such that it
    can be used for a subsequent attempt.

Arguments:

    Graph - Supplies a pointer to the graph instance to reset.

Return Value:

    PH_S_CONTINUE_GRAPH_SOLVING - Graph was successfully reset and graph solving
        should continue.

    PH_S_GRAPH_SOLVING_STOPPED - Graph solving has been stopped.  The graph is
        not reset and solving should not continue.

    PH_S_TABLE_RESIZE_IMMINENT - The reset was not performed as a table resize
        is imminent (and thus, attempts at solving this current graph can be
        stopped).

    PH_E_SYSTEM_CALL_FAILED - A system call failed.

--*/
{
    HRESULT Result;
    PGRAPH_INFO Info;
    CU_STREAM Stream;
    CU_RESULT CuResult;
    ULONG TotalNumberOfPages;
    ULONG TotalNumberOfLargePages;
    ULONG TotalNumberOfCacheLines;
    PASSIGNED_MEMORY_COVERAGE Coverage;
    PASSIGNED_PAGE_COUNT NumberOfAssignedPerPage;
    PASSIGNED_LARGE_PAGE_COUNT NumberOfAssignedPerLargePage;
    PASSIGNED_CACHE_LINE_COUNT NumberOfAssignedPerCacheLine;

    //
    // Initialize aliases.
    //

    Info = Graph->CuGraphInfo;

    Result = PH_S_CONTINUE_GRAPH_SOLVING;

    Stream = Graph->CuKernelContext->Streams.Reset;

    //
    // Clear scalar values.
    //

    Graph->Collisions = 0;
    Graph->NumberOfEmptyVertices = 0;
    Graph->DeletedEdgeCount = 0;
    Graph->VisitedVerticesCount = 0;

    Graph->TraversalDepth = 0;
    Graph->TotalTraversals = 0;
    Graph->MaximumTraversalDepth = 0;

    Graph->Flags.Shrinking = FALSE;
    Graph->Flags.IsAcyclic = FALSE;

    Graph->AddKeysElapsedCycles.QuadPart = 0;
    Graph->HashKeysElapsedCycles.QuadPart = 0;
    Graph->AddHashedKeysElapsedCycles.QuadPart = 0;

    //
    // XXX: temp.
    //

    return Result;

    //
    // Clear the bitmap buffers.
    //

#define ZERO_BITMAP_BUFFER(Name)                           \
    ASSERT(0 == Info->Name##BufferSizeInBytes -          \
           ((Info->Name##BufferSizeInBytes >> 3) << 3)); \
    CU_ZERO(Graph->Name.Buffer,                        \
            Info->Name##BufferSizeInBytes,               \
            Stream)

    ZERO_BITMAP_BUFFER(DeletedEdgesBitmap);
    ZERO_BITMAP_BUFFER(VisitedVerticesBitmap);
    ZERO_BITMAP_BUFFER(AssignedBitmap);
    ZERO_BITMAP_BUFFER(IndexBitmap);

    //
    // "Empty" all of the nodes.
    //

#define EMPTY_ARRAY(Name)                                           \
    ASSERT(0 == Info->Name##SizeInBytes -                         \
           ((Info->Name##SizeInBytes >> 3) << 3));                \
    CU_MEMSET(Graph->Name, 0xffffffff, Info->Name##SizeInBytes, Stream)

    EMPTY_ARRAY(First);
    EMPTY_ARRAY(Next);
    EMPTY_ARRAY(Edges);

    //
    // Avoid the overhead of resetting the memory coverage if we're in "first
    // graph wins" mode and have been requested to skip memory coverage.
    //

    if (!FindBestGraph(Graph)) {
        goto End;
    }

    //
    // Clear the assigned memory coverage counts and arrays.
    //

    Coverage = &Graph->AssignedMemoryCoverage;

    //
    // Capture the totals and pointers prior to zeroing the struct.
    //

    TotalNumberOfPages = Coverage->TotalNumberOfPages;
    TotalNumberOfLargePages = Coverage->TotalNumberOfLargePages;
    TotalNumberOfCacheLines = Coverage->TotalNumberOfCacheLines;

    NumberOfAssignedPerPage = Coverage->NumberOfAssignedPerPage;
    NumberOfAssignedPerLargePage = Coverage->NumberOfAssignedPerLargePage;
    NumberOfAssignedPerCacheLine = Coverage->NumberOfAssignedPerCacheLine;

    CU_ZERO(Coverage, sizeof(*Coverage), Stream);

    //
    // Restore the totals and pointers.
    //

    Coverage->TotalNumberOfPages = TotalNumberOfPages;
    Coverage->TotalNumberOfLargePages = TotalNumberOfLargePages;
    Coverage->TotalNumberOfCacheLines = TotalNumberOfCacheLines;

    Coverage->NumberOfAssignedPerPage = NumberOfAssignedPerPage;
    Coverage->NumberOfAssignedPerLargePage = NumberOfAssignedPerLargePage;
    Coverage->NumberOfAssignedPerCacheLine = NumberOfAssignedPerCacheLine;

#define ZERO_ASSIGNED_ARRAY(Name) \
    CU_ZERO(Coverage->Name, Info->Name##SizeInBytes, Stream)

    //ZERO_ASSIGNED_ARRAY(NumberOfAssignedPerPage);
    //ZERO_ASSIGNED_ARRAY(NumberOfAssignedPerLargePage);
    //ZERO_ASSIGNED_ARRAY(NumberOfAssignedPerCacheLine);

    //
    // We're done, finish up.
    //

    goto End;

Error:

    if (SUCCEEDED(Result)) {
        Result = E_UNEXPECTED;
    }

    //
    // Intentional follow-on to End.
    //

End:

    return Result;
}

EXTERN_C
DEVICE
BOOLEAN
GraphCuIsAcyclic(
    _In_ PGRAPH Graph
    )
{
    return FALSE;
}

#if 0
EXTERN_C
FORCEINLINE
DEVICE
EDGE
AbsoluteEdge(
    _In_ PGRAPH Graph,
    _In_ EDGE Edge,
    _In_ ULONG Index
    )
{
    ULONG AbsEdge;
    ULONG MaskedEdge;

    MaskedEdge = Edge & Graph->EdgeMask;

    AbsEdge = (MaskedEdge + (Index * Graph->NumberOfEdges));
    return AbsEdge;
}
#endif

EXTERN_C
KERNEL
VOID
GraphCuAssignKernel(
    _In_ PGRAPH Graph
    )
/*++

Routine Description:

    This routine is called after a graph has determined to be acyclic.  It is
    responsible for walking the graph and assigning values to edges in order to
    complete the perfect hash solution.

Arguments:

    Graph - Supplies a pointer to the graph to operate on.

Return Value:

    None.

--*/
{

}

EXTERN_C
DEVICE
VOID
GraphCuCalculateAssignedMemoryCoverage(
    _In_ PGRAPH Graph
    )
{

}

EXTERN_C
DEVICE
VOID
GraphCuCalculateAssignedMemoryCoverageForKeysSubset(
    _In_ PGRAPH Graph
    )
{

}

EXTERN_C
DEVICE
HRESULT
GraphCuRegisterSolved(
    _In_ PGRAPH Graph,
    _Inout_ PGRAPH *NewGraphPointer
    )
{
    return PH_S_STOP_GRAPH_SOLVING;
}


EXTERN_C
DEVICE
HRESULT
GraphCuSolve(
    _In_ PGRAPH Graph,
    _Out_ PGRAPH *NewGraphPointer
    )
/*++

Routine Description:

    Add all keys to the hypergraph using the unique seeds to hash each key into
    two vertex values, connected by a "hyper-edge".  Determine if the graph is
    acyclic, if it is, we've "solved" the graph.  If not, we haven't.

Arguments:

    Graph - Supplies a pointer to the graph to be solved.

    NewGraphPointer - Supplies the address of a variable which will receive the
        address of a new graph instance to be used for solving if the routine
        returns PH_S_USE_NEW_GRAPH_FOR_SOLVING.

Return Value:

    PH_S_STOP_GRAPH_SOLVING - Stop graph solving.

    PH_S_GRAPH_SOLVING_STOPPED - Graph solving has been stopped.

    PH_S_CONTINUE_GRAPH_SOLVING - Continue graph solving.

    PH_S_USE_NEW_GRAPH_FOR_SOLVING - Continue graph solving but use the graph
        returned via the NewGraphPointer parameter.

--*/
{
    PKEY Keys;
    HRESULT Result;
    CU_RESULT CuResult;
    ULONG NumberOfKeys;
    ULONG BlocksPerGrid;
    ULONG ThreadsPerBlock;
    ULONG SharedMemoryInBytes;
    PCU_KERNEL_CONTEXT Ctx;
    PCU_KERNEL_STREAMS Streams;
    PASSIGNED_MEMORY_COVERAGE Coverage;

    //
    // Initialize aliases.
    //

    Ctx = Graph->CuKernelContext;
    Streams = &Ctx->Streams;
    Keys = (PKEY)Graph->DeviceKeys;
    NumberOfKeys = Graph->NumberOfKeys;
    BlocksPerGrid = Graph->CuBlocksPerGrid;
    ThreadsPerBlock = Graph->CuThreadsPerBlock;

    //
    // Attempt to add all the keys to the graph.
    //

    Graph->CuHashKeysResult = E_FAIL;

    SharedMemoryInBytes = (

        //
        // Account for the GRAPH_SHARED structure.
        //

        sizeof(GRAPH_SHARED) +

        //
        // Account for the array of result codes (one per block) for HashKeys.
        //

        (sizeof(HRESULT) * BlocksPerGrid)

    );

    //
    // Launch the hash kernel.
    //

    Graph->CuHashKeysResult = S_OK;
    Graph->CuIsAcyclicResult = S_OK;

    SharedMemoryInBytes = 0;

    HashAllMultiplyShiftRKernel<<<
        BlocksPerGrid,
        ThreadsPerBlock,
        SharedMemoryInBytes,
        Streams->Solve
    >>>(
        Keys,
        Graph->NumberOfEdges,
        NumberOfKeys,
        Graph->VertexPairs,
        Graph->SortedVertexPairsDevice,
        Graph->VertexPairsIndexDevice,
        Graph->First,
        Graph->Next,
        Graph->Edges,
        Graph->VertexMask,
        Graph->Seeds,
        &Graph->CuHashKeysResult
    );

    CuResult = hipDeviceSynchronize();
    CU_CHECK(CuResult, hipDeviceSynchronize);

    Result = Graph->CuHashKeysResult;
    if (FAILED(Result)) {
        if (Result == PH_E_GRAPH_VERTEX_COLLISION_FAILURE) {
            //printf("Collided!\n");
            Graph->CuVertexCollisionFailures++;
            goto Failed;
        }
        PH_ERROR(GraphCuSolve_AddKeys, Result);
        Result = PH_S_STOP_GRAPH_SOLVING;
        goto End;
    }

    Graph->CuNoVertexCollisionFailures++;

    //
    // When we've got more than 225,000 keys, launching the acyclic *and* the
    // sort kernel at the same time yields a memory allocation error in the sort
    // kernel.  In this case, just issue a device sync between the kernel calls.
    //

    if (NumberOfKeys > 225000) {

        GraphCuIsAcyclicKernel<<<1, 1, 0, Streams->IsAcyclic>>>(
            NumberOfKeys,
            thrust::device_ptr<VERTEX_PAIR>(Graph->VertexPairs),
            &Graph->CuIsAcyclicResult
        );

        CuResult = hipDeviceSynchronize();
        CU_CHECK(CuResult, hipDeviceSynchronize);

        GraphCuSortVertexPairsKernel<<<1, 1, 0, Streams->SortVertexPairs>>>(
            NumberOfKeys,
            thrust::device_ptr<VERTEX_PAIR>(Graph->SortedVertexPairsDevice),
            thrust::device_ptr<ULONG>(Graph->VertexPairsIndexDevice)
        );


    } else {

        //
        // We can launch both kernels and let them execute concurrently.
        //

        GraphCuIsAcyclicKernel<<<1, 1, 0, Streams->IsAcyclic>>>(
            NumberOfKeys,
            thrust::device_ptr<VERTEX_PAIR>(Graph->VertexPairs),
            &Graph->CuIsAcyclicResult
        );

        GraphCuSortVertexPairsKernel<<<1, 1, 0, Streams->SortVertexPairs>>>(
            NumberOfKeys,
            thrust::device_ptr<VERTEX_PAIR>(Graph->SortedVertexPairsDevice),
            thrust::device_ptr<ULONG>(Graph->VertexPairsIndexDevice)
        );

    }

    CuResult = hipDeviceSynchronize();
    CU_CHECK(CuResult, hipDeviceSynchronize);

    Result = Graph->CuIsAcyclicResult;
    if (FAILED(Result)) {
        if (Result == PH_E_GRAPH_CYCLIC_FAILURE) {
            Graph->CuCyclicGraphFailures++;
            goto Failed;
        }
        PH_ERROR(GraphCuSolve_IsAcyclic, Result);
        Result = PH_S_STOP_GRAPH_SOLVING;
        goto End;
    }

    //
    // We created an acyclic graph.
    //

    Graph->CuFinishedCount++;

    //
    // Launch the assignment kernel.
    //

    SharedMemoryInBytes = 0;

    GraphCuAssignKernel<<<
        BlocksPerGrid,
        ThreadsPerBlock,
        SharedMemoryInBytes,
        Streams->Assign
    >>>(Graph);

    //
    // If we're in "first graph wins" mode and we reach this point, optionally
    // calculate coverage and then finish up.
    //

    if (FirstSolvedGraphWins(Graph)) {
        if (WantsAssignedMemoryCoverage(Graph)) {
            GraphCuCalculateAssignedMemoryCoverage(Graph);
        }
        Result = PH_S_STOP_GRAPH_SOLVING;
        goto End;
    }

    //
    // If we reach this mode, we're in "find best graph" mode, so, register the
    // solved graph then continue solving.
    //

    ASSERT(FindBestGraph(Graph));

    //
    // Calculate memory coverage information if applicable.
    //

    if (WantsAssignedMemoryCoverage(Graph)) {
        GraphCuCalculateAssignedMemoryCoverage(Graph);
    } else if (WantsAssignedMemoryCoverageForKeysSubset(Graph)) {
        GraphCuCalculateAssignedMemoryCoverageForKeysSubset(Graph);
    }

    //
    // This is a bit hacky; the graph traversal depth is proving to be more
    // interesting than initially thought, such that we've recently added a
    // best coverage type predicate aimed at maximizing it, which means we
    // need to make the value available from the coverage struct in order for
    // the X-macro to work, which means we're unnecessarily duplicating the
    // value at the table and coverage level.  Not particularly elegant.
    //

    Coverage = &Graph->AssignedMemoryCoverage;
    Coverage->MaxGraphTraversalDepth = Graph->MaximumTraversalDepth;

    //
    // Ditto for total traversals, empty vertices and collisions.
    //

    Coverage->TotalGraphTraversals = Graph->TotalTraversals;
    Coverage->NumberOfEmptyVertices = Graph->NumberOfEmptyVertices;
    Coverage->NumberOfCollisionsDuringAssignment = Graph->Collisions;

    //
    // Register the solved graph.  We can return this result directly.
    //

    Result = GraphCuRegisterSolved(Graph, NewGraphPointer);

    //
    // Intentional follow-on to End.
    //

End:

    //
    // Intentional follow-on to Error.
    //

Error:
    return Result;

Failed:
    Graph->CuFailedAttempts++;
    return PH_S_CONTINUE_GRAPH_SOLVING;
}

EXTERN_C
DEVICE
HRESULT
GraphCuCreateKernelContext(
    _In_ PGRAPH Graph
    )
{
    HRESULT Result;
    CU_RESULT CuResult;
    PCU_STREAM Stream;
    PCU_STREAM FirstStream;
    PCU_STREAM LastStream;
    PCU_KERNEL_STREAMS Streams;
    PCU_KERNEL_CONTEXT Ctx;

    if (Graph->CuKernelContext != NULL) {
        Result = S_FALSE;
        goto End;
    }

    CuResult = hipMalloc(&Graph->CuKernelContext, sizeof(*Ctx));
    CU_CHECK(CuResult, hipMalloc);

    //
    // Create streams.
    //

    Ctx = Graph->CuKernelContext;
    Streams = &Ctx->Streams;
    FirstStream = &Streams->FirstStream;
    LastStream = &Streams->LastStream;
    for (Stream = FirstStream; Stream <= LastStream; Stream++) {
        CREATE_STREAM(Stream);
    }

    //
    // Initialize our random state.
    //

    hiprand_init(Graph->CuRngSeed,
                Graph->CuRngSubsequence,
                Graph->CuRngOffset,
                &Ctx->RngState.Philox4);

    Result = S_OK;
    goto End;

Error:

    if (SUCCEEDED(Result)) {
        Result = E_UNEXPECTED;
    }

    //
    // Intentional follow-on to End.
    //

End:

    return Result;
}


EXTERN_C
GLOBAL
VOID
PerfectHashCudaEnterSolvingLoop(
    _In_ PGRAPH Graph
    )
/*++

Routine Description:

    This is the main entry point for the CUDA graph solving implementation.
    This kernel is intended to be called with a single thread.  It launches
    child kernels dynamically.

Arguments:

    Graph - Supplies a pointer to a GRAPH structure for which solving is to be
        performed.

Return Value:

    N.B. The return value is provided to the caller via Graph->CuKernelResult.

    S_OK - Success.

--*/
{
    HRESULT Result;
    PGRAPH NewGraph;

    //
    // Abort if the kernel is called with more than one thread.
    //

    if (GridDim.x > 1  || GridDim.y > 1  || GridDim.z > 1 ||
        BlockDim.x > 1 || BlockDim.y > 1 || BlockDim.z > 1)
    {
        Result = PH_E_CU_KERNEL_SOLVE_LOOP_INVALID_DIMENSIONS;
        goto End;
    }

    //ASSERT(Graph->SizeOfStruct == sizeof(GRAPH));
    if (Graph->SizeOfStruct != sizeof(GRAPH)) {
        printf("%u != %u!\n", (ULONG)Graph->SizeOfStruct, (ULONG)sizeof(GRAPH));
        return;
    }

    if (Graph->CuKernelContext == NULL) {
        Result = GraphCuCreateKernelContext(Graph);
        if (FAILED(Result)) {
            PH_ERROR(GraphCuCreateKernelContext, Result);
            return;
        }
        printf("Created context successfully.\n");
    }

    //
    // Begin the solving loop.
    //

    do {

        if (!GraphCuShouldWeContinueTryingToSolve(Graph, &Result)) {
            break;
        }

        Result = GraphCuLoadNewSeeds(Graph);
        if (FAILED(Result)) {
            break;
        }

        Result = GraphCuReset(Graph);
        if (FAILED(Result)) {
            break;
        } else if (Result != PH_S_CONTINUE_GRAPH_SOLVING) {
            break;
        }

        NewGraph = NULL;
        Result = GraphCuSolve(Graph, &NewGraph);
        if (FAILED(Result)) {
            break;
        }

        if (Result == PH_S_STOP_GRAPH_SOLVING ||
            Result == PH_S_GRAPH_SOLVING_STOPPED) {
            ASSERT(NewGraph == NULL);
            break;
        }

        if (Result == PH_S_USE_NEW_GRAPH_FOR_SOLVING) {
            ASSERT(NewGraph != NULL);
            Graph = NewGraph;
        } else {

            //
            // Invariant check: result should be PH_S_CONTINUE_GRAPH_SOLVING
            // at this point.
            //

            ASSERT(Result == PH_S_CONTINUE_GRAPH_SOLVING);
        }

        //
        // Continue the loop and attempt another solve.
        //

    } while (TRUE);

    //
    // We're done, finish up.
    //

    goto End;

#if 0
Error:

    if (SUCCEEDED(Result)) {
        Result = E_UNEXPECTED;
    }

    //
    // Intentional follow-on to End.
    //

#endif
End:
    Graph->CuKernelResult = Result;
}

// vim:set ts=8 sw=4 sts=4 tw=80 expandtab filetype=cuda formatoptions=croql   :
